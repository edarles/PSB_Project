#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* 
    Marching cubes

    This sample extracts a geometric isosurface from a volume dataset using
    the marching cubes algorithm. It uses the scan (prefix sum) function from
    the CUDPP library to perform stream compaction. Similar techniques can
    be used for other problems that require a variable-sized output per
    thread.

    For more information on marching cubes see:
    http://local.wasp.uwa.edu.au/~pbourke/geometry/polygonise/
    http://en.wikipedia.org/wiki/Marching_cubes

    Volume data courtesy:
    http://www9.informatik.uni-erlangen.de/External/vollib/

    For more information on CUDPP ("CUDA Data Parallel Primitives Library"):
    http://www.gpgpu.org/developer/cudpp

    The algorithm consists of several stages:

    1. Execute "classifyVoxel" kernel
    This evaluates the volume at the corners of each voxel and computes the
    number of vertices each voxel will generate.
    It is executed using one thread per voxel.
    It writes two arrays - voxelOccupied and voxelVertices to global memory.
    voxelOccupied is a flag indicating if the voxel is non-empty.

    2. Scan "voxelOccupied" array (using CUDPP cudppScan())
    Read back the total number of occupied voxels from GPU to CPU.
    This is the sum of the last value of the exclusive scan and the last
    input value.

    3. Execute "compactVoxels" kernel
    This compacts the voxelOccupied array to get rid of empty voxels.
    This allows us to run the complex "generateTriangles" kernel on only
    the occupied voxels.

    4. Scan voxelVertices array
    This gives the start address for the vertex data for each voxel.
    We read back the total number of vertices generated from GPU to CPU.

    Note that by using a custom scan function we could combine the above two
    scan operations above into a single operation.

    5. Execute "generateTriangles" kernel
    This runs only on the occupied voxels.
    It looks up the field values again and generates the triangle data,
    using the results of the scan to write the output to the correct addresses.
    The marching cubes look-up tables are stored in 1D textures.

    6. Render geometry
    Using number of vertices from readback.
*/

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <GL/glew.h>
#include <cutil.h>
#include <cuda_gl_interop.h>

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#define SKIP_EMPTY_VOXELS 1

// kernels
#include "marchingCubes_kernel.cu"
#include "tables.h"
#include "cudpp/cudpp.h"

// constants
const unsigned int window_width = 512;
const unsigned int window_height = 512;

char *volumeFilename = "Bucky.raw";

uint3 gridSizeLog2 = make_uint3(5, 5, 5);
uint3 gridSizeShift;
uint3 gridSize;
uint3 gridSizeMask;

float3 voxelSize;
uint numVoxels = 0;
uint maxVerts = 0;
uint activeVoxels = 0;
uint totalVerts = 0;

float isoValue = 0.2f;
float dIsoValue = 0.005f;

CUDPPHandle scanplan;

// device data
GLuint posVbo, normalVbo;

uchar *d_volume = 0;
uint *d_voxelVerts = 0;
uint *d_voxelVertsScan = 0;
uint *d_voxelOccupied = 0;
uint *d_voxelOccupiedScan = 0;
uint *d_compactedVoxelArray;

// tables
uint* d_numVertsTable = 0;
uint* d_edgeTable = 0;
uint* d_triTable = 0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float3 rotate = make_float3(0.0, 0.0, 0.0);
float3 translate= make_float3(0.0, 0.0, -3.0);

// toggles
bool wireframe = false;
bool animate = false;
bool lighting = true;
bool render = true;
bool compute = true;

// forward declarations
void runTest(int argc, char** argv);

CUTBoolean initGL();
void createVBO(GLuint* vbo, unsigned int size);
void deleteVBO(GLuint* vbo);

void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void idle();
void reshape(int w, int h);

void computeIsosurface();
void dumpBuffer(uint *d_buffer, int nelements);
void mainMenu(int i);

////////////////////////////////////////////////////////////////////////////////
// Load raw data from disk
////////////////////////////////////////////////////////////////////////////////
uchar *loadRawFile(char *filename, int size)
{
	FILE *fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Error opening file '%s'\n", filename);
        return 0;
    }

	uchar *data = (uchar *) malloc(size);
	size_t read = fread(data, 1, size, fp);
	fclose(fp);

    printf("Read '%s', %d bytes\n", filename, read);

    return data;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    runTest(argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
// initialize marching cubes
////////////////////////////////////////////////////////////////////////////////
void
initMC(int argc, char** argv)
{
    // parse command line arguments
    int n;
    if (cutGetCmdLineArgumenti( argc, (const char**) argv, "grid", &n)) {
        gridSizeLog2.x = gridSizeLog2.y = gridSizeLog2.z = n;
    }
    if (cutGetCmdLineArgumenti( argc, (const char**) argv, "gridx", &n)) {
        gridSizeLog2.x = n;
    }
    if (cutGetCmdLineArgumenti( argc, (const char**) argv, "gridy", &n)) {
        gridSizeLog2.y = n;
    }
    if (cutGetCmdLineArgumenti( argc, (const char**) argv, "gridz", &n)) {
        gridSizeLog2.z = n;
    }

    char *filename;
    if (cutGetCmdLineArgumentstr( argc, (const char**) argv, "file", &filename)) {
        volumeFilename = filename;
    }

    gridSize = make_uint3(1<<gridSizeLog2.x, 1<<gridSizeLog2.y, 1<<gridSizeLog2.z);
    gridSizeMask = make_uint3(gridSize.x-1, gridSize.y-1, gridSize.z-1);
    gridSizeShift = make_uint3(0, gridSizeLog2.x, gridSizeLog2.x+gridSizeLog2.y);

    numVoxels = gridSize.x*gridSize.y*gridSize.z;
    voxelSize = make_float3(2.0f / gridSize.x, 2.0f / gridSize.y, 2.0f / gridSize.z);
    maxVerts = gridSize.x*gridSize.y*100;

    printf("grid: %d x %d x %d = %d voxels\n", gridSize.x, gridSize.y, gridSize.z, numVoxels);
    printf("max verts = %d\n", maxVerts);

#if SAMPLE_VOLUME
    // load volume data
    char* path = cutFindFilePath(volumeFilename, argv[0]);
    if (path == 0) {
        fprintf(stderr, "Error finding file '%s'\n", volumeFilename);
        exit(EXIT_FAILURE);
    }

    int size = gridSize.x*gridSize.y*gridSize.z*sizeof(uchar);
    uchar *volume = loadRawFile(path, size);
    CUDA_SAFE_CALL(hipMalloc((void**) &d_volume, size));
    CUDA_SAFE_CALL(hipMemcpy(d_volume, volume, size, hipMemcpyHostToDevice) );
    free(volume);

    // bind to linear texture
    CUDA_SAFE_CALL(hipBindTexture(0, volumeTex, d_volume, hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned)));
#endif

    // create VBOs
    createVBO(&posVbo, maxVerts*sizeof(float)*4);
    CUDA_SAFE_CALL(cudaGLRegisterBufferObject(posVbo));

    createVBO(&normalVbo, maxVerts*sizeof(float)*4);
    CUDA_SAFE_CALL(cudaGLRegisterBufferObject(normalVbo));

    // allocate textures
    CUDA_SAFE_CALL(hipMalloc((void**) &d_edgeTable, 256*sizeof(uint)));
    CUDA_SAFE_CALL(hipMemcpy(d_edgeTable, edgeTable, 256*sizeof(uint), hipMemcpyHostToDevice) );
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
    CUDA_SAFE_CALL(hipBindTexture(0, edgeTex, d_edgeTable, channelDesc) );

    CUDA_SAFE_CALL(hipMalloc((void**) &d_triTable, 256*16*sizeof(uint)));
    CUDA_SAFE_CALL(hipMemcpy(d_triTable, triTable, 256*16*sizeof(uint), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL(hipBindTexture(0, triTex, d_triTable, channelDesc) );

    CUDA_SAFE_CALL(hipMalloc((void**) &d_numVertsTable, 256*sizeof(uint)));
    CUDA_SAFE_CALL(hipMemcpy(d_numVertsTable, numVertsTable, 256*sizeof(uint), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL(hipBindTexture(0, numVertsTex, d_numVertsTable, channelDesc) );

    // allocate device memory
    unsigned int memSize = sizeof(uint) * numVoxels;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_voxelVerts, memSize));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_voxelVertsScan, memSize));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_voxelOccupied, memSize));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_voxelOccupiedScan, memSize));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_compactedVoxelArray, memSize));

    // initialize CUDPP scan
    CUDPPConfiguration config;
    config.algorithm = CUDPP_SCAN;
    config.datatype = CUDPP_UINT;
    config.op = CUDPP_ADD;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
    cudppPlan(&scanplan, config, numVoxels, 1, 0);
}

void cleanup()
{
    deleteVBO(&posVbo);
    deleteVBO(&normalVbo);

    if (d_volume)
        CUDA_SAFE_CALL(hipFree(d_volume));
    CUDA_SAFE_CALL(hipFree(d_edgeTable));
    CUDA_SAFE_CALL(hipFree(d_triTable));
    CUDA_SAFE_CALL(hipFree(d_numVertsTable));

    CUDA_SAFE_CALL(hipFree(d_voxelVerts));
    CUDA_SAFE_CALL(hipFree(d_voxelVertsScan));
    CUDA_SAFE_CALL(hipFree(d_voxelOccupied));
    CUDA_SAFE_CALL(hipFree(d_voxelOccupiedScan));
    CUDA_SAFE_CALL(hipFree(d_compactedVoxelArray));

    cudppDestroyPlan(scanplan);
}

void initMenus()
{
    glutCreateMenu(mainMenu);
    glutAddMenuEntry("Toggle animation [ ]", ' ');
    glutAddMenuEntry("Increment isovalue [+]", '+');
    glutAddMenuEntry("Decrement isovalue [-]", '-');
    glutAddMenuEntry("Toggle computation [c]", 'c');
    glutAddMenuEntry("Toggle rendering [r]", 'r');
    glutAddMenuEntry("Toggle lighting [l]", 'l');
    glutAddMenuEntry("Toggle wireframe [w]", 'w');
    glutAddMenuEntry("Quit (esc)", '\033');
    glutAttachMenu(GLUT_RIGHT_BUTTON);
}

void
runTest(int argc, char** argv)
{
    CUT_DEVICE_INIT(argc, argv);

    // Create GL context
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda Marching Cubes");

    // initialize GL
    if(CUTFalse == initGL()) {
        return;
    }

    // register callbacks
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutIdleFunc(idle);
    glutReshapeFunc(reshape);
    initMenus();

    initMC(argc, argv);

    // start rendering mainloop
    glutMainLoop();
}

void dumpBuffer(uint *d_buffer, int nelements)
{
    uint bytes = nelements*sizeof(uint);
    uint *h_buffer = (uint *) malloc(bytes);
    CUDA_SAFE_CALL(hipMemcpy(h_buffer, d_buffer, bytes, hipMemcpyDeviceToHost) );
    for(int i=0; i<nelements; i++) {
        printf("%d: %u\n", i, h_buffer[i]);
    }
    printf("\n");
    free(h_buffer);
}

#define DEBUG_BUFFERS 0

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void
computeIsosurface()
{
    int threads = 128;
    dim3 grid(numVoxels / threads, 1, 1);
    // get around maximum grid size of 65535 in each dimension
    if (grid.x > 65535) {
        grid.y = grid.x / 32768;
        grid.x = 32768;
    }

    // calculate number of vertices need per voxel
    classifyVoxel<<<grid, threads>>>(d_voxelVerts, d_voxelOccupied, d_volume, 
                                     gridSize, gridSizeShift, gridSizeMask, 
                                     numVoxels, voxelSize, isoValue);
    CUT_CHECK_ERROR("classifyVoxel failed");
#if DEBUG_BUFFERS
    printf("voxelVerts:\n");
    dumpBuffer(d_voxelVerts, numVoxels);
#endif

#if SKIP_EMPTY_VOXELS

    // scan voxel occupied array
    cudppScan(scanplan, d_voxelOccupiedScan, d_voxelOccupied, numVoxels);
#if DEBUG_BUFFERS
    printf("voxelOccupiedScan:\n");
    dumpBuffer(d_voxelOccupiedScan, numVoxels);
#endif

    // read back values to calculate total number of non-empty voxels
    // since we are using an exclusive scan, the total is the last value of
    // the scan result plus the last value in the input array
    {
        uint lastElement, lastScanElement;
        CUDA_SAFE_CALL(hipMemcpy((void *) &lastElement, 
                       (void *) (d_voxelOccupied + numVoxels-1), 
                       sizeof(uint), hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipMemcpy((void *) &lastScanElement, 
                       (void *) (d_voxelOccupiedScan + numVoxels-1), 
                       sizeof(uint), hipMemcpyDeviceToHost));
        activeVoxels = lastElement + lastScanElement;
    }

    if (activeVoxels==0) {
        // return if there are no full voxels
        totalVerts = 0;
        return;
    }

    // compact voxel index array
    compactVoxels<<<grid, threads>>>(d_compactedVoxelArray, d_voxelOccupied, 
                                     d_voxelOccupiedScan, numVoxels);
    CUT_CHECK_ERROR("compactVoxels failed");

#endif // SKIP_EMPTY_VOXELS

    // scan voxel vertex count array
    cudppScan(scanplan, d_voxelVertsScan, d_voxelVerts, numVoxels);
#if DEBUG_BUFFERS
    printf("voxelVertsScan:\n");
    dumpBuffer(d_voxelVertsScan, numVoxels);
#endif

    // readback total number of vertices
    {
        uint lastElement, lastScanElement;
        CUDA_SAFE_CALL(hipMemcpy((void *) &lastElement, 
                       (void *) (d_voxelVerts + numVoxels-1), 
                       sizeof(uint), hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipMemcpy((void *) &lastScanElement, 
                       (void *) (d_voxelVertsScan + numVoxels-1), 
                       sizeof(uint), hipMemcpyDeviceToHost));
        totalVerts = lastElement + lastScanElement;
    }

    // generate triangles, writing to vertex buffers
    float4 *d_pos = 0, *d_normal = 0;
    CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&d_pos, posVbo));
    CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&d_normal, normalVbo));

#if SKIP_EMPTY_VOXELS
    dim3 grid2((int) ceil(activeVoxels / (float) NTHREADS), 1, 1);
#else
    dim3 grid2((int) ceil(numVoxels / (float) NTHREADS), 1, 1);
#endif
    while(grid2.x > 65535) {
        grid2.x/=2;
        grid2.y*=2;
    }
#if SAMPLE_VOLUME
    generateTriangles2<<<grid2, NTHREADS>>>(d_pos, d_normal, 
                                            d_compactedVoxelArray, 
                                            d_voxelVertsScan, d_volume, 
                                            gridSize, gridSizeShift, 
                                            gridSizeMask, voxelSize, 
                                            isoValue, activeVoxels, 
                                            maxVerts);
#else
    generateTriangles<<<grid2, NTHREADS>>>(d_pos, d_normal, 
                                           d_compactedVoxelArray, 
                                           d_voxelVertsScan, gridSize, 
                                           gridSizeShift, gridSizeMask, 
                                           voxelSize, isoValue, 
                                           activeVoxels, maxVerts);
#endif
    CUT_CHECK_ERROR("generateTriangles failed");

    CUDA_SAFE_CALL(cudaGLUnmapBufferObject(normalVbo));
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject(posVbo));
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize OpenGL
////////////////////////////////////////////////////////////////////////////////
CUTBoolean
initGL()
{
    // initialize necessary OpenGL extensions
    glewInit();
    if (! glewIsSupported("GL_VERSION_2_0 " 
		                  )) {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return CUTFalse;
    }

    // default initialization
    glClearColor(0.1, 0.2, 0.3, 1.0);
    glEnable(GL_DEPTH_TEST);

    // good old-fashioned fixed function lighting
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float white[] = { 1.0, 1.0, 1.0, 1.0 };
    float ambient[] = { 0.1, 0.1, 0.1, 1.0 };
    float diffuse[] = { 0.9, 0.9, 0.9, 1.0 };
    float lightPos[] = { 0.0, 0.0, 1.0, 0.0 };

    glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, ambient);
    glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse);
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, black);

    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, white);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModelfv(GL_LIGHT_MODEL_AMBIENT, black);

    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    
    glutReportErrors();

    return CUTTrue;
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void
createVBO(GLuint* vbo, unsigned int size)
{
    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    glutReportErrors();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void
deleteVBO(GLuint* vbo)
{
    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);
    CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(*vbo));

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
// Render isosurface geometry from the vertex buffers
////////////////////////////////////////////////////////////////////////////////
void renderIsosurface()
{
    glBindBuffer(GL_ARRAY_BUFFER, posVbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);
    glEnableClientState(GL_VERTEX_ARRAY);

    glBindBufferARB(GL_ARRAY_BUFFER_ARB, normalVbo);
    glNormalPointer(GL_FLOAT, sizeof(float)*4, 0);
    glEnableClientState(GL_NORMAL_ARRAY);

    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_TRIANGLES, 0, totalVerts);
    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_NORMAL_ARRAY);

    glBindBuffer(GL_ARRAY_BUFFER, 0);
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void
display()
{
    // run CUDA kernel to generate geometry
    if (compute) {
        computeIsosurface();
    }

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(translate.x, translate.y, translate.z);
    glRotatef(rotate.x, 1.0, 0.0, 0.0);
    glRotatef(rotate.y, 0.0, 1.0, 0.0);

    glPolygonMode(GL_FRONT_AND_BACK, wireframe? GL_LINE : GL_FILL);
    if (lighting) {
        glEnable(GL_LIGHTING);
    }

    // render
    if (render) {
        glPushMatrix();
        glRotatef(180.0, 0.0, 1.0, 0.0);
        glRotatef(90.0, 1.0, 0.0, 0.0);
        renderIsosurface();
        glPopMatrix();
    }

    glDisable(GL_LIGHTING);

    glutSwapBuffers();
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void
keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch(key) {
    case(27) :
        cleanup();
        exit(0);
    case '=':
        isoValue += 0.01;
        break;
    case '-':
        isoValue -= 0.01;
        break;
    case '+':
        isoValue += 0.1;
        break;
    case '_':
        isoValue -= 0.1;
        break;
    case 'w':
        wireframe = !wireframe;
        break;
    case ' ':
        animate = !animate;
        break;
    case 'l':
        lighting = !lighting;
        break;
    case 'r':
        render = !render;
        break;
    case 'c':
        compute = !compute;
        break;
    }

    printf("isoValue = %f\n", isoValue);
    printf("voxels = %d\n", activeVoxels);
    printf("verts = %d\n", totalVerts);
    printf("occupancy: %d / %d = %.2f%%\n", 
           activeVoxels, numVoxels, activeVoxels*100.0f / (float) numVoxels);

    if (!compute) {
        computeIsosurface();        
    }

    glutPostRedisplay();
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void
mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN) {
        mouse_buttons |= 1<<button;
    } else if (state == GLUT_UP) {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

void motion(int x, int y)
{
    float dx = x - mouse_old_x;
    float dy = y - mouse_old_y;

    if (mouse_buttons==1) {
        rotate.x += dy * 0.2;
        rotate.y += dx * 0.2;
    } else if (mouse_buttons==2) {
        translate.x += dx * 0.01;
        translate.y -= dy * 0.01;
    } else if (mouse_buttons==3) {
        translate.z += dy * 0.01;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

void idle()
{
    if (animate) {
        isoValue += dIsoValue;
        if (isoValue < 0.1f || isoValue > 0.9f)
            dIsoValue *= -1.0f;
    }
    glutPostRedisplay();
}

void reshape(int w, int h)
{
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (float) w / (float) h, 0.1, 10.0);

    glMatrixMode(GL_MODELVIEW);
    glViewport(0, 0, w, h);
}

void mainMenu(int i)
{
    keyboard((unsigned char) i, 0, 0);
}
