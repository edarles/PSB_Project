#include "hip/hip_runtime.h"
/*
* Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws.
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
* OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
* OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
* OR PERFORMANCE OF THIS SOURCE CODE.
*
* U.S. Government End Users.  This source code is a "commercial item" as
* that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
* "commercial computer software" and "commercial computer software
* documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
* and is provided to the U.S. Government only as a commercial end item.
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
* source code with only those rights set forth herein.
*/

/*
    Volume rendering sample

    This sample loads a 3D volume from disk and displays it using
    ray marching and 3D textures.
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <GL/glew.h>

#if defined (__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#include <cuda_gl_interop.h>
#include <cutil.h>

typedef unsigned int uint;
typedef unsigned char uchar;

#include <volumeRender_kernel.cu>

char *volumeFilename = "Bucky.raw";
hipExtent volumeSize = make_hipExtent(32, 32, 32);

uint width = 512, height = 512;
dim3 blockSize(16, 16);
dim3 gridSize(width / blockSize.x, height / blockSize.y);

float3 viewRotation;
float3 viewTranslation = make_float3(0.0, 0.0, -4.0f);
float invViewMatrix[12];

float density = 0.05f;
float brightness = 1.0f;
float transferOffset = 0.0f;
float transferScale = 1.0f;
bool linearFiltering = true;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;
GLuint pbo = 0;     // OpenGL pixel buffer object

void initPixelBuffer();

// render image using CUDA
void render()
{
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeof(float4)*3) );

    // map PBO to get CUDA device pointer
    uint *d_output;
    CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&d_output, pbo));

    CUDA_SAFE_CALL(hipMemset(d_output, 0, width*height*4));

    // call CUDA kernel, writing results to PBO
    d_render<<<gridSize, blockSize>>>(d_output, width, height, density, brightness, transferOffset, transferScale);
    CUT_CHECK_ERROR("kernel failed");

    CUDA_SAFE_CALL(cudaGLUnmapBufferObject(pbo));
}

// display results using OpenGL (called by GLUT)
void display()
{
    // use OpenGL to build view matrix
    GLfloat modelView[16];
    glMatrixMode(GL_MODELVIEW);
    glPushMatrix();
        glLoadIdentity();
        glRotatef(-viewRotation.x, 1.0, 0.0, 0.0);
        glRotatef(-viewRotation.y, 0.0, 1.0, 0.0);
        glTranslatef(-viewTranslation.x, -viewTranslation.y, -viewTranslation.z);
    glGetFloatv(GL_MODELVIEW_MATRIX, modelView);
    glPopMatrix();

    invViewMatrix[0] = modelView[0]; invViewMatrix[1] = modelView[4]; invViewMatrix[2] = modelView[8]; invViewMatrix[3] = modelView[12];
    invViewMatrix[4] = modelView[1]; invViewMatrix[5] = modelView[5]; invViewMatrix[6] = modelView[9]; invViewMatrix[7] = modelView[13];
    invViewMatrix[8] = modelView[2]; invViewMatrix[9] = modelView[6]; invViewMatrix[10] = modelView[10]; invViewMatrix[11] = modelView[14];

    render();

    // display results
    glClear(GL_COLOR_BUFFER_BIT);

    // draw image from PBO
    glDisable(GL_DEPTH_TEST);
    glRasterPos2i(0, 0);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    glutSwapBuffers();
    glutReportErrors();
}

void idle()
{
}

void keyboard(unsigned char key, int x, int y)
{
    switch(key) {
        case 27:
            exit(0);
            break;
        case 'f':
            linearFiltering = !linearFiltering;
            tex.filterMode = linearFiltering ? hipFilterModeLinear : hipFilterModePoint;
            break;
        case '=':
            density += 0.01;
            break;
        case '-':
            density -= 0.01;
            break;
        case '+':
            density += 0.1;
            break;
        case '_':
            density -= 0.1;
            break;

        case ']':
            brightness += 0.1;
            break;
        case '[':
            brightness -= 0.1;
            break;

        case ';':
            transferOffset += 0.01;
            break;
        case '\'':
            transferOffset -= 0.01;
            break;

        case '.':
            transferScale += 0.01;
            break;
        case ',':
            transferScale -= 0.01;
            break;

        default:
            break;
    }
    printf("density = %.2f, brightness = %.2f, transferOffset = %.2f, transferScale = %.2f\n", density, brightness, transferOffset, transferScale);
    glutPostRedisplay();
}

int ox, oy;
int buttonState = 0;

void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
        buttonState |= 1<<button;
    else if (state == GLUT_UP)
        buttonState = 0;

    ox = x; oy = y;
    glutPostRedisplay();
}

void motion(int x, int y)
{
    float dx, dy;
    dx = x - ox;
    dy = y - oy;

    if (buttonState == 3) {
        // left+middle = zoom
        viewTranslation.z += dy / 100.0;
    } 
    else if (buttonState & 2) {
        // middle = translate
        viewTranslation.x += dx / 100.0;
        viewTranslation.y -= dy / 100.0;
    }
    else if (buttonState & 1) {
        // left = rotate
        viewRotation.x += dy / 5.0;
        viewRotation.y += dx / 5.0;
    }

    ox = x; oy = y;
    glutPostRedisplay();
}

void reshape(int x, int y)
{
    width = x; height = y;
    initPixelBuffer();

    glViewport(0, 0, x, y);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0); 
}

void initCuda(uchar *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize) );

    // The sample does not work with pageable memory
    // This is a known issue for beta that will be fixed for the public release
#define USE_PAGE_LOCKED_MEMORY 1
#if USE_PAGE_LOCKED_MEMORY
    // copy to page-locked mem
    hipPitchedPtr pagelockedPtr;
    pagelockedPtr.pitch = volumeSize.width*sizeof(uchar);
    pagelockedPtr.xsize = volumeSize.width;
    pagelockedPtr.ysize = volumeSize.height;
    size_t size = volumeSize.width*volumeSize.height*volumeSize.depth*sizeof(uchar);
    CUDA_SAFE_CALL( hipHostMalloc(&(pagelockedPtr.ptr), size) );
    memcpy(pagelockedPtr.ptr, h_volume, size);
#endif

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
#if USE_PAGE_LOCKED_MEMORY
    copyParams.srcPtr   = pagelockedPtr;
#else
    copyParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
#endif
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );  

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeClamp;  // wrap texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;

    // bind array to 3D texture
    CUDA_SAFE_CALL(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

    // create transfer function texture
    float4 transferFunc[] = {
        {  0.0, 0.0, 0.0, 0.0, },
        {  1.0, 0.0, 0.0, 1.0, },
        {  1.0, 0.5, 0.0, 1.0, },
        {  1.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 1.0, 1.0, },
        {  0.0, 0.0, 1.0, 1.0, },
        {  1.0, 0.0, 1.0, 1.0, },
        {  0.0, 0.0, 0.0, 0.0, },
    };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray* d_transferFuncArray;
    CUDA_SAFE_CALL(hipMallocArray( &d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1)); 
    CUDA_SAFE_CALL(hipMemcpyToArray( d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    CUDA_SAFE_CALL( hipBindTextureToArray( transferTex, d_transferFuncArray, channelDesc2));
}

void cleanup()
{
    CUDA_SAFE_CALL(hipFreeArray(d_volumeArray));
    CUDA_SAFE_CALL(hipFreeArray(d_transferFuncArray));
	CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(pbo));    
	glDeleteBuffersARB(1, &pbo);
}

int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

void initPixelBuffer()
{
    if (pbo) {
        // delete old buffer
        CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(pbo));
        glDeleteBuffersARB(1, &pbo);
    }

    // create pixel buffer object for display
    glGenBuffersARB(1, &pbo);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
	glBufferDataARB(GL_PIXEL_UNPACK_BUFFER_ARB, width*height*sizeof(GLubyte)*4, 0, GL_STREAM_DRAW_ARB);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

	CUDA_SAFE_CALL(cudaGLRegisterBufferObject(pbo));

    // calculate new grid size
    gridSize = dim3(iDivUp(width, blockSize.x), iDivUp(height, blockSize.y));
}

// Load raw data from disk
uchar *loadRawFile(char *filename, size_t size)
{
	FILE *fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Error opening file '%s'\n", filename);
        return 0;
    }

	uchar *data = (uchar *) malloc(size);
	size_t read = fread(data, 1, size, fp);
	fclose(fp);

    printf("Read '%s', %d bytes\n", filename, read);

    return data;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    CUT_DEVICE_INIT(argc, argv);

    // parse arguments
    char *filename;
    if (cutGetCmdLineArgumentstr( argc, (const char**) argv, "file", &filename)) {
        volumeFilename = filename;
    }
    int n;
    if (cutGetCmdLineArgumenti( argc, (const char**) argv, "size", &n)) {
        volumeSize.width = volumeSize.height = volumeSize.depth = n;
    }
    if (cutGetCmdLineArgumenti( argc, (const char**) argv, "xsize", &n)) {
        volumeSize.width = n;
    }
    if (cutGetCmdLineArgumenti( argc, (const char**) argv, "ysize", &n)) {
        volumeSize.height = n;
    }
    if (cutGetCmdLineArgumenti( argc, (const char**) argv, "zsize", &n)) {
         volumeSize.depth = n;
    }

    // load volume data
    char* path = cutFindFilePath(volumeFilename, argv[0]);
    if (path == 0) {
        fprintf(stderr, "Error finding file '%s'\n", volumeFilename);
        exit(EXIT_FAILURE);
    }

    size_t size = volumeSize.width*volumeSize.height*volumeSize.depth;
    uchar *h_volume = loadRawFile(path, size);
    initCuda(h_volume, volumeSize);
    free(h_volume);

    printf("Press '=' and '-' to change density\n"
           "      ']' and '[' to change brightness\n"
           "      ';' and ''' to modify transfer function offset\n"
           "      '.' and ',' to modify transfer function scale\n");

    // initialize GLUT callback functions
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
    glutInitWindowSize(width, height);
    glutCreateWindow("CUDA volume rendering");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutReshapeFunc(reshape);
    glutIdleFunc(idle);

    glewInit();
    if (!glewIsSupported("GL_VERSION_2_0 GL_ARB_pixel_buffer_object")) {
        fprintf(stderr, "Required OpenGL extensions missing.");
        exit(-1);
    }
    initPixelBuffer();

    atexit(cleanup);

    glutMainLoop();
    return 0;
}
