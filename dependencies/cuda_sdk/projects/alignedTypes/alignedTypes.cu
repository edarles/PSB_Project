#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/*
 * This is a simple test showing huge access speed gap
 * between aligned and misaligned structures
 * (those having/missing __align__ keyword).
 * It measures per-element copy throughput for 
 * aligned and misaligned structures on 
 * big chunks of data.
 */



#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil.h>



////////////////////////////////////////////////////////////////////////////////
// Misaligned types
////////////////////////////////////////////////////////////////////////////////
typedef unsigned char uint8;

typedef unsigned short int uint16;

typedef struct{
    unsigned char r, g, b, a;
} RGBA8_misaligned;

typedef struct{
    unsigned int l, a;
} LA32_misaligned;

typedef struct{
    unsigned int r, g, b;
} RGB32_misaligned;

typedef struct{
    unsigned int r, g, b, a;
} RGBA32_misaligned;



////////////////////////////////////////////////////////////////////////////////
// Aligned types
////////////////////////////////////////////////////////////////////////////////
typedef struct __align__(4){
    unsigned char r, g, b, a;
} RGBA8;

typedef unsigned int I32;

typedef struct __align__(8){
    unsigned int l, a;
} LA32;

typedef struct __align__(16){
    unsigned int r, g, b;
} RGB32;

typedef struct __align__(16){
    unsigned int r, g, b, a;
} RGBA32;


////////////////////////////////////////////////////////////////////////////////
// Because G80 class hardware natively supports global memory operations
// only with data elements of 4, 8 and 16 bytes, if structure size
// exceeds 16 bytes, it can't be efficiently read or written,
// since more than one global memory non-coalescable load/store instructions
// will be generated, even if __align__ option is supplied.
// "Structure of arrays" storage strategy offers best performance 
// in general case. See section 5.1.2 of the Programming Guide.
////////////////////////////////////////////////////////////////////////////////
typedef struct __align__(16){
    RGBA32 c1, c2;
} RGBA32_2;



////////////////////////////////////////////////////////////////////////////////
// Common host and device functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Round a / b to nearest lower integer value
int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b){
    return (a % b != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
int iAlignDown(int a, int b){
    return a - a % b;
}



////////////////////////////////////////////////////////////////////////////////
// Simple CUDA kernel.
// Copy is carried out on per-element basis,
// so it's not per-byte in case of padded structures.
////////////////////////////////////////////////////////////////////////////////
template<class TData> __global__ void testKernel(
    TData *d_odata,
    TData *d_idata,
    int numElements
){
    const int        tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    for(int pos = tid; pos < numElements; pos += numThreads)
        d_odata[pos] = d_idata[pos];
}



////////////////////////////////////////////////////////////////////////////////
// Validation routine for simple copy kernel.
// We must know "packed" size of TData (number_of_fields * sizeof(simple_type))
// and compare only these "packed" parts of the structure, 
// containig actual user data. The compiler behavior with padding bytes
// is undefined, since padding is merely a placeholder 
// and doesn't contain any user data.
////////////////////////////////////////////////////////////////////////////////
template<class TData> int testCPU(
    TData *h_odata,
    TData *h_idata,
    int numElements,
    int packedElementSize
){
    for(int pos = 0; pos < numElements; pos++){
        TData src = h_idata[pos];
        TData dst = h_odata[pos];
        for(int i = 0; i < packedElementSize; i++)
            if( ((char *)&src)[i] != ((char *)&dst)[i] ) return 0;
    }

    return 1;
}



////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//Memory chunk size in bytes. Reused for test
#ifdef __DEVICE_EMULATION__
    const int       MEM_SIZE = 16000000;
    const int NUM_ITERATIONS = 1;
#else
    const int       MEM_SIZE = 50000000;
    const int NUM_ITERATIONS = 32;
#endif

//GPU input and output data
unsigned char *d_idata, *d_odata;
//CPU input data and instance of GPU output data
unsigned char *h_idataCPU, *h_odataGPU;
unsigned int hTimer;



template<class TData> void runTest(int packedElementSize){
    const int totalMemSizeAligned = iAlignDown(MEM_SIZE, sizeof(TData));
    const int         numElements = iDivDown(MEM_SIZE, sizeof(TData));

    //Clean output buffer before current test
    CUDA_SAFE_CALL( hipMemset(d_odata, 0, MEM_SIZE) );
    //Run test
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutResetTimer(hTimer) );
    CUT_SAFE_CALL( cutStartTimer(hTimer) );
    for(int i = 0; i < NUM_ITERATIONS; i++){
        testKernel<<<64, 256>>>(
            (TData *)d_odata,
            (TData *)d_idata,
            numElements
        );
        CUT_CHECK_ERROR("testKernel() execution failed\n");
    }
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutStopTimer(hTimer) );
    double gpuTime = cutGetTimerValue(hTimer) / NUM_ITERATIONS;
    printf(
        "Avg. time: %f ms / Copy throughput: %f GB/s.\n", gpuTime,
        (double)totalMemSizeAligned / (gpuTime * 0.001 * 1073741824.0)
    );

    //Read back GPU results and run validation
    CUDA_SAFE_CALL( hipMemcpy(h_odataGPU, d_odata, MEM_SIZE, hipMemcpyDeviceToHost) );
    int flag = testCPU(
        (TData *)h_odataGPU,
        (TData *)h_idataCPU,
        numElements,
        packedElementSize
    );
    printf(flag ? "TEST PASSED\n" : "TEST FAILED\n" );
}



int main(int argc, char **argv){
    int i;

    CUT_DEVICE_INIT(argc, argv);
    CUT_SAFE_CALL( cutCreateTimer(&hTimer) );

    printf("Allocating memory...\n");
        h_idataCPU = (unsigned char *)malloc(MEM_SIZE);
        h_odataGPU = (unsigned char *)malloc(MEM_SIZE);
        CUDA_SAFE_CALL(hipMalloc((void **)&d_idata, MEM_SIZE));
        CUDA_SAFE_CALL(hipMalloc((void **)&d_odata, MEM_SIZE));

    printf("Generating host input data array...\n");
        for(i = 0; i < MEM_SIZE; i++)
            h_idataCPU[i] = (i & 0xFF) + 1;
    printf("Uploading input data to GPU memory...\n");
        CUDA_SAFE_CALL(hipMemcpy(d_idata, h_idataCPU, MEM_SIZE, hipMemcpyHostToDevice) );


    printf("Testing misaligned types...\n");
        printf("uint8...\n");
        runTest<uint8>(1);

        printf("uint16...\n");
        runTest<uint16>(2);

        printf("RGBA8_misaligned...\n");
        runTest<RGBA8_misaligned>(4);

        printf("LA32_misaligned...\n");
        runTest<LA32_misaligned>(8);

        printf("RGB32_misaligned...\n");
        runTest<RGB32_misaligned>(12);

        printf("RGBA32_misaligned...\n");
        runTest<RGBA32_misaligned>(16);

    printf("Testing aligned types...\n");
        printf("RGBA8...\n");
        runTest<RGBA8>(4);

        printf("I32...\n");
        runTest<I32>(4);

        printf("LA32...\n");
        runTest<LA32>(8);

        printf("RGB32...\n");
        runTest<RGB32>(12);

        printf("RGBA32...\n");
        runTest<RGBA32>(16);

        printf("RGBA32_2...\n");
        runTest<RGBA32_2>(32);


    printf("Shutting down...\n");
        CUDA_SAFE_CALL(hipFree(d_idata));
        CUDA_SAFE_CALL(hipFree(d_odata));
        free(h_odataGPU);
        free(h_idataCPU);

    CUT_SAFE_CALL( cutDeleteTimer(hTimer) );
    CUT_EXIT(argc, argv);
}
