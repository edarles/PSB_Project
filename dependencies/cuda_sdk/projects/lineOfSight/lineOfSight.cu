#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

// This sample is an implementation of a simple line-of-sight algorithm:
// Given a height map and a ray originating at some observation point,
// it computes all the points along the ray that are visible from the
// observation point.
// It is based on the description made in "Guy E. Blelloch. Vector models
// for data-parallel computing. MIT Press, 1990" and uses the parallel scan
// primitive provided by the CUDPP library (http://www.gpgpu.org/developer/cudpp/).

#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, library
#include "cudpp/cudpp.h"

// includes, project
#include "cutil.h"
#include "cutil_math.h"

////////////////////////////////////////////////////////////////////////////////
// declaration, types

// Boolean
typedef unsigned char Bool;
enum {
    False = 0,
    True = 1
};

// 2D height field
struct HeightField {
	int     width;
    float*  height;
};

// Ray
struct Ray {
	float3 origin;
    float2 dir;
    int    length;
    float  oneOverLength;
};

////////////////////////////////////////////////////////////////////////////////
// declaration, variables

// Height field texture reference
texture<float, 2, hipReadModeElementType> g_HeightFieldTex;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);
__global__ void computeAngles_kernel(const Ray, float*);
__global__ void computeVisibilities_kernel(const float*, const float*, int, Bool*);
void lineOfSight_gold(const HeightField, const Ray, Bool*);
__device__ __host__ float2 getLocation(const Ray, int);
__device__ __host__ float getAngle(const Ray, float2, float);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a line-of-sight test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv) 
{
    ////////////////////////////////////////////////////////////////////////////
    // Device initialization
    CUT_DEVICE_INIT(argc, argv);
    
    ////////////////////////////////////////////////////////////////////////////
    // Timer
    
    // Create
    uint timer;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    
    // Number of iterations to get accurate timing
#ifdef __DEVICE_EMULATION__
    uint numIterations = 1;
#else
    uint numIterations = 100;
#endif

    ////////////////////////////////////////////////////////////////////////////
    // Height field
    
    HeightField heightField;
    
    // Allocate in host memory
    int2 dim = make_int2(10000, 100);
    heightField.width = dim.x;
    int heightFieldSize = dim.x * dim.y * sizeof(float);
    CUT_SAFE_MALLOC(heightField.height = (float*)malloc(heightFieldSize));
    
    // Fill in with an arbitrary sine surface
    for (int x = 0; x < dim.x; ++x)
        for (int y = 0; y < dim.y; ++y) {
            float amp = 0.1f * (x + y);
            float period = 2.0f + amp;
            *(heightField.height + dim.x * y + x) =
                amp * (sinf(sqrtf((float)(x * x + y * y)) * 2.0f * 3.1416f / period) + 1.0f);
        }
    
    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
               hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* heightFieldArray;
    CUDA_SAFE_CALL(hipMallocArray(&heightFieldArray, &channelDesc, dim.x, dim.y));
     
    // Initialize device memory
    CUDA_SAFE_CALL(hipMemcpyToArray(heightFieldArray, 0, 0, heightField.height,
                                     heightFieldSize, hipMemcpyHostToDevice));

    // Set texture parameters
    g_HeightFieldTex.addressMode[0] = hipAddressModeClamp;
    g_HeightFieldTex.addressMode[1] = hipAddressModeClamp;
    g_HeightFieldTex.filterMode = hipFilterModePoint;
    g_HeightFieldTex.normalized = 0;

    // Bind CUDA array to texture reference
    CUDA_SAFE_CALL(hipBindTextureToArray(g_HeightFieldTex, heightFieldArray,
                                          channelDesc));
                                          
    ////////////////////////////////////////////////////////////////////////////
    // Ray (starts at origin and traverses the height field diagonally)
    
    Ray ray;
    ray.origin = make_float3(0, 0, 2.0f);
    int2 dir = make_int2(dim.x - 1, dim.y - 1); 
    ray.dir = make_float2((float)dir.x, (float)dir.y);
    ray.length = max(abs(dir.x), abs(dir.y));
    ray.oneOverLength = 1.0f / ray.length;

    ////////////////////////////////////////////////////////////////////////////
    // View angles

    // Allocate view angles for each point along the ray
    float* d_angles;
    int raySize = ray.length * sizeof(float);
    CUDA_SAFE_CALL(hipMalloc((void**)&d_angles, raySize));
    
    // Allocate result of max-scan operation on the array of view angles
    float* d_scannedAngles;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_scannedAngles, raySize));
    
    ////////////////////////////////////////////////////////////////////////////
    // Visibility results

    // Allocate visibility results for each point along the ray
    Bool* d_visibilities;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_visibilities, raySize));
    Bool* h_visibilities;
    CUT_SAFE_MALLOC(h_visibilities = (Bool*)malloc(raySize));
    Bool* h_visibilitiesRef;
    CUT_SAFE_MALLOC(h_visibilitiesRef = (Bool*)malloc(raySize));
    
    ////////////////////////////////////////////////////////////////////////////
    // Reference solution
    lineOfSight_gold(heightField, ray, h_visibilitiesRef);
    
    ////////////////////////////////////////////////////////////////////////////
    // Device solution

    // Execution configuration
    dim3 block(256);
    dim3 grid((uint)ceil(ray.length / (double)block.x));
    
    // Scan configuration
    CUDPPHandle scanPlan;
   
    CUDPPConfiguration config;
    config.algorithm = CUDPP_SCAN;
    config.op = CUDPP_MAX;
    config.datatype = CUDPP_FLOAT;  
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
    cudppPlan(&scanPlan, config, ray.length, 1, 0);
    
    // Compute device solution
    printf("Line of sight\n");
    cutStartTimer(timer);
    for (uint i = 0; i < numIterations; ++i) {
    
        // Compute view angle for each point along the ray
        computeAngles_kernel<<<grid, block>>>(ray, d_angles);
        CUT_CHECK_ERROR("Kernel execution failed");
        
        // Perform a max-scan operation on the array of view angles
        cudppScan(scanPlan, d_scannedAngles, d_angles, ray.length);
        CUT_CHECK_ERROR("Kernel execution failed");
        
        // Compute visibility results based on the array of view angles
        // and its scanned version
        computeVisibilities_kernel<<<grid, block>>>(d_angles, d_scannedAngles,
                                                    ray.length, d_visibilities);
        CUT_CHECK_ERROR("Kernel execution failed");
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    cudppDestroyPlan(scanPlan);
    CUT_CHECK_ERROR("Kernel execution failed");

    // Copy visibility results back to the host
    CUDA_SAFE_CALL(hipMemcpy(h_visibilities, d_visibilities, raySize,
                              hipMemcpyDeviceToHost));
                              
    // Compare device visibility results against reference results
    CUTBoolean res = cutCompareub(h_visibilitiesRef, h_visibilities, ray.length);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);
    cutResetTimer(timer);

    // Cleanup memory
    free(heightField.height);
    free(h_visibilities);
    free(h_visibilitiesRef);
    CUDA_SAFE_CALL(hipFree(d_angles));
    CUDA_SAFE_CALL(hipFree(d_scannedAngles));
    CUDA_SAFE_CALL(hipFree(d_visibilities));
    CUDA_SAFE_CALL(hipFreeArray(heightFieldArray));
}

////////////////////////////////////////////////////////////////////////////////
//! Compute view angles for each point along the ray
//! @param ray         ray
//! @param angles      view angles
////////////////////////////////////////////////////////////////////////////////
__global__ void computeAngles_kernel(const Ray ray, float* angles)
{
    uint i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < ray.length) {
        float2 location = getLocation(ray, i + 1);
        float height = tex2D(g_HeightFieldTex, location.x, location.y);
        float angle = getAngle(ray, location, height);
        angles[i] = angle;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute visibility for each point along the ray
//! @param angles          view angles
//! @param scannedAngles   max-scanned view angles
//! @param numAngles       number of view angles
//! @param visibilities    boolean array indicating the visibility of each point
//!                        along the ray
////////////////////////////////////////////////////////////////////////////////
__global__ void computeVisibilities_kernel(const float* angles,
                                           const float* scannedAngles,
                                           int numAngles,
                                           Bool* visibilities)
{
    uint i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numAngles)
        visibilities[i] = scannedAngles[i] <= angles[i];
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set
//! @param heightField     height field
//! @param ray             ray
//! @param visibilities    boolean array indicating the visibility of each point
//!                        along the ray
////////////////////////////////////////////////////////////////////////////////
void lineOfSight_gold(const HeightField heightField, const Ray ray,
                      Bool* visibilities)
{
    float angleMax = asinf(-1.0f);
    for (int i = 0; i < ray.length; ++i) {
        float2 location = getLocation(ray, i + 1);
        float height = *(heightField.height
                       + heightField.width * (int)floorf(location.y)
                       + (int)floorf(location.x));
        float angle = getAngle(ray, location, height);
        if (angle > angleMax) {
            angleMax = angle;
            visibilities[i] = True;
        }
        else
            visibilities[i] = False;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute the 2D coordinates of the point located at i steps from the origin
//! of the ray
//! @param ray      ray
//! @param i        integer offset along the ray
////////////////////////////////////////////////////////////////////////////////
__device__ __host__ float2 getLocation(const Ray ray, int i)
{
    float step = i * ray.oneOverLength;
    return make_float2(ray.origin.x, ray.origin.y) + step * ray.dir;
}

////////////////////////////////////////////////////////////////////////////////
//! Compute the angle of view between a 3D point and the origin of the ray
//! @param ray        ray
//! @param location   2D coordinates of the input point
//! @param height     height of the input point
////////////////////////////////////////////////////////////////////////////////
__device__ __host__ float getAngle(const Ray ray, float2 location, float height)
{
    float2 dir = location - make_float2(ray.origin.x, ray.origin.y);
    return atanf((height - ray.origin.z) / length(dir));
}
