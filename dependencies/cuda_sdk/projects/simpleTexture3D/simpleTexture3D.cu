/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/*
    3D texture sample

    This sample loads a 3D volume from disk and displays slices through it
    using 3D texture lookups.
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <GL/glew.h>

#if defined (__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#include <cuda_gl_interop.h>
#include <cutil.h>

typedef unsigned int uint;
typedef unsigned char uchar;

#include "simpleTexture3D_kernel.cu"

const char *volumeFilename = "Bucky.raw";
const hipExtent volumeSize = make_hipExtent(32, 32, 32);

const uint width = 512, height = 512;
const dim3 blockSize(16, 16, 1);
const dim3 gridSize(width / blockSize.x, height / blockSize.y);

float w = 0.5;  // texture coordinate in z

hipArray *d_volumeArray = 0;
GLuint pbo;     // OpenGL pixel buffer object

bool linearFiltering = true;
bool animate = true;

// render image using CUDA
void render()
{
    // map PBO to get CUDA device pointer
    uint *d_output;
    CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&d_output, pbo));

    // call CUDA kernel, writing results to PBO
    d_render<<<gridSize, blockSize>>>(d_output, width, height, w);
    CUT_CHECK_ERROR("kernel failed");

    CUDA_SAFE_CALL(cudaGLUnmapBufferObject(pbo));
}

// display results using OpenGL (called by GLUT)
void display()
{
    render();

    // display results
    glClear(GL_COLOR_BUFFER_BIT);

    // draw image from PBO
    glDisable(GL_DEPTH_TEST);
    glRasterPos2i(0, 0);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    glutSwapBuffers();
    glutReportErrors();
}

void idle()
{
    if (animate) {
        w += 0.01f;
        glutPostRedisplay();
    }
}

void keyboard(unsigned char key, int x, int y)
{
    switch(key) {
        case 27:
            exit(0);
            break;
        case '=':
        case '+':
            w += 0.01;
            break;
        case '-':
            w -= 0.01;
            break;
        case 'f':
            linearFiltering = !linearFiltering;
            tex.filterMode = linearFiltering ? hipFilterModeLinear : hipFilterModePoint;
            break;
        case ' ':
            animate = !animate;
            break;
        default:
            break;
    }
    glutPostRedisplay();
}

void reshape(int x, int y)
{
    glViewport(0, 0, x, y);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0); 
}

void initCuda(const uchar *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize) );

    // The sample does not work with pageable memory
    // This is a known issue for beta that will be fixed for the public release
#define USE_PAGE_LOCKED_MEMORY 1
#if USE_PAGE_LOCKED_MEMORY
    // copy to page-locked mem
    hipPitchedPtr pagelockedPtr;
    pagelockedPtr.pitch = volumeSize.width*sizeof(uchar);
    pagelockedPtr.xsize = volumeSize.width;
    pagelockedPtr.ysize = volumeSize.height;
    size_t size = volumeSize.width*volumeSize.height*volumeSize.depth*sizeof(uchar);
    CUDA_SAFE_CALL( hipHostMalloc(&(pagelockedPtr.ptr), size) );
    memcpy(pagelockedPtr.ptr, h_volume, size);
#endif

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
#if USE_PAGE_LOCKED_MEMORY
    copyParams.srcPtr   = pagelockedPtr;
#else
    copyParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
#endif
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    tex.addressMode[1] = hipAddressModeWrap;
    tex.addressMode[2] = hipAddressModeWrap;

    // bind array to 3D texture
    CUDA_SAFE_CALL(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
}

void cleanup()
{
	CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(pbo));    
	glDeleteBuffersARB(1, &pbo);
}

void initOpenGL()
{
    // create pixel buffer object
    glGenBuffersARB(1, &pbo);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
	glBufferDataARB(GL_PIXEL_UNPACK_BUFFER_ARB, width*height*sizeof(GLubyte)*4, 0, GL_STREAM_DRAW_ARB);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

	CUDA_SAFE_CALL(cudaGLRegisterBufferObject(pbo));
}

// Load raw data from disk
uchar *loadRawFile(const char *filename, size_t size)
{
	FILE *fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Error opening file '%s'\n", filename);
        return 0;
    }

	uchar *data = (uchar *) malloc(size);
	size_t read = fread(data, 1, size, fp);
	fclose(fp);

    printf("Read '%s', %d bytes\n", filename, read);

    return data;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    CUT_DEVICE_INIT(argc, argv);

    // load volume data
    const char* path = cutFindFilePath(volumeFilename, argv[0]);
    if (path == 0) {
        fprintf(stderr, "Error finding file '%s'\n", volumeFilename);
        exit(EXIT_FAILURE);
    }

    size_t size = volumeSize.width*volumeSize.height*volumeSize.depth;
    uchar *h_volume = loadRawFile(path, size);
    initCuda(h_volume, volumeSize);
    free(h_volume);

    printf("Press space to toggle animation\n"
           "Press '+' and '-' to change displayed slice\n");

    // initialize GLUT callback functions
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
    glutInitWindowSize(width, height);
    glutCreateWindow("CUDA 3D texture");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutReshapeFunc(reshape);
    glutIdleFunc(idle);

    glewInit();
    if (!glewIsSupported("GL_VERSION_2_0 GL_ARB_pixel_buffer_object")) {
        fprintf(stderr, "Required OpenGL extensions missing.");
        exit(-1);
    }
    initOpenGL();

    atexit(cleanup);

    glutMainLoop();
    return 0;
}
