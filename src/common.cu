#include "hip/hip_runtime.h"

#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

extern "C"
{
  int getCudaEnabledDeviceCount()
{
    int count;
    hipError_t error = hipGetDeviceCount( &count );
    if (error == hipErrorNoDevice)
    {
	printf("NO DEVICE\n");
        count = 0;
    }
    else if (error == hipErrorInsufficientDriver)
    {
        count = -1;
    }
    else //should never happen
    {
        checkCudaErrors(error);
    }
    return count;
  }

  bool cudaInit(int argc, char **argv)
  {
       int count = getCudaEnabledDeviceCount();
       printf("count:%d\n",count);

       int devID = findCudaGLDevice(argc, (const char **)argv);
       
       if (devID < 0)
       {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
	    return false;
        }
	else {
		 hipDeviceProp_t deviceProp;
		 checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
       		 printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
		 return true;
	}
    }

    void cudaGLInit(int argc, char **argv)
    {
        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        findCudaGLDevice(argc, (const char **)argv);
    }
 
    void allocateArray(void **devPtr, size_t size)
    {
        hipMalloc(devPtr, size);
    }

    void freeArray(void *devPtr)
    {
        hipFree(devPtr);
    }

    void threadSync()
    {
        hipDeviceSynchronize();
    }

    void copyArrayToDevice(void *device, const void *host, int offset, int size)
    {
        hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice);
    }

    void copyArrayDeviceToDevice(void *device, const void *host, int offset, int size)
    {
        hipMemcpy((char *) device + offset, host, size, hipMemcpyDeviceToDevice);
    }

    void copyArrayFromDevice(void *host, const void *device,
                             struct hipGraphicsResource **cuda_vbo_resource, int size)
    {
        hipMemcpy(host, device, size, hipMemcpyDeviceToHost);

    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint &numBlocks, uint &numThreads)
    {
	uint blockSize = 32;
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }
}
