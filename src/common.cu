#include "hip/hip_runtime.h"

#define CUDA

#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <common.cuh>

#if defined (CUDA)
	#include <hip/hip_runtime_api.h>
	#include <hip/hip_runtime.h>
	#include <>
	#include <cuda_gl_interop.h>
	#include <hip/hip_runtime_api.h>
	#include <helper_functions.h>
	#include <hip/hip_runtime_api.h>
	#include <helper_cuda_gl.h>
	#include <hip/hip_runtime.h>
#endif

extern "C"
{

  #if defined (CUDA)
  int getCudaEnabledDeviceCount()
  {
    int count;
    hipDeviceSynchronize();
    hipError_t error = hipGetDeviceCount( &count );
    if (error == hipErrorNoDevice)
    {
	printf("NO DEVICE\n");
        count = 0;
    }
    else if (error == hipErrorInsufficientDriver)
    {
        count = -1;
    }
   // else //should never happen
    {
        checkCudaErrors(error);
    }
    return count;
  }

  bool cudaInit(int argc, char **argv)
  {
       int count = getCudaEnabledDeviceCount();

       int devID = findCudaGLDevice(argc, (const char **)argv);
       
       if (devID < 0)
       {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
	    return false;
        }
	else {
		 hipDeviceProp_t deviceProp;
		 checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
       		 printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
		 return true;
	}
    }

    void cudaGLInit(int argc, char **argv)
    {
        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        findCudaGLDevice(argc, (const char **)argv);
    }
 
    void allocateArray(void **devPtr, size_t size)
    {
        hipMalloc(devPtr, size);
    }

    void freeArray(void *devPtr)
    {
        hipFree(devPtr);
    }

    void threadSync()
    {
        hipDeviceSynchronize();
    }

    void copyArrayToDevice(void *device, const void *host, int offset, int size)
    {
        hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice);
    }

    void copyArrayDeviceToDevice(void *device, const void *host, int offset, int size)
    {
        hipMemcpy((char *) device + offset, host, size, hipMemcpyDeviceToDevice);
    }

    void copyArrayDeviceToDevice2(void *device, const void *host, int offset, int size)
    {
        hipMemcpy(device, (char *) host + offset, size, hipMemcpyDeviceToDevice);
    }

    void copyArrayFromDevice(void *device, const void *host, int offset, int size)
    {
        hipMemcpy((char *) device + offset, host, size, hipMemcpyDeviceToHost);
    }
    int iDivUp(int a, int b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(int n, int &numBlocks, int &numThreads)
    {
	int blockSize = 512;
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
	//printf("nbBlocks:%d nbThreads:%d\n",numBlocks,numThreads);
    }
    #endif
}
