#include "hip/hip_runtime.h"
#include <PeriodicHeightFieldCollision_Kernel.cuh>
#include <double3.h>
/******************************************************************************************/
/******************************************************************************************/
extern "C"
{
/******************************************************************************************/
/******************************************************************************************/
__host__ __device__ double  calculateHeight_Periodic(double3 pos, uint nbFunc, double* A, double* k, double* theta, double* phi)
{
	double y = 0;
	for(uint i=0;i<nbFunc;i++)
		y+= A[i]*cos(k[i]*(pos.x*cos(theta[i])+pos.z*sin(theta[i]))+phi[i]);
	return y;
}
/******************************************************************************************/
/******************************************************************************************/
__host__ __device__ double3 approximateNormale_Periodic(double3 pos, uint nbFunc, double* A, double* k, double* theta, double* phi)
{
	double d=0.01;
	double3 V1 = make_double3(pos.x+d,pos.y,pos.z);
	double3 V2 = make_double3(pos.x-d,pos.y,pos.z);
	double dpx = (calculateHeight_Periodic(V1,nbFunc,A,k,theta,phi)-calculateHeight_Periodic(V2,nbFunc,A,k,theta,phi))/(2*d);
	V1.x = pos.x; V1.z = pos.z+d; V2.x = pos.x; V2.z = pos.z-d;
	double dpy = 1.0;
	double dpz = (calculateHeight_Periodic(V1,nbFunc,A,k,theta,phi)-calculateHeight_Periodic(V2,nbFunc,A,k,theta,phi))/(2*d);
	double norme = sqrt(dpx*dpx + dpy*dpy + dpz*dpz);
	double3 N;
	N.x =-dpx/norme;
	N.y = dpy/norme;
	N.z =-dpz/norme;
	return N;
}

/******************************************************************************************/
/******************************************************************************************/
__global__ void collisionSystem_Periodic_HeightFieldCollision_Kernel
	   (double3* newPos, double3 *newVel, double radiusParticle, float dt, uint nbBodiesP, 
	    uint nbFunc, double* A, double* k, double* theta, double* phi,
	    float3 min_, float3 max_, float elast)
{
	
	int indexP = blockIdx.x * blockDim.x + threadIdx.x;
    	if(indexP < nbBodiesP){
		double3 pos = newPos[indexP];
		if(pos.x>=min_.x && pos.z>=min_.z && pos.x<=max_.x && pos.z<=max_.z){
			double y = calculateHeight_Periodic(pos,nbFunc,A,k,theta,phi) + min_.y;
			if(y>=pos.y){
				double3 pInter = make_double3(pos.x,y,pos.z);
				float r = 0;
				if(length(newVel[indexP])>0 && elast>0 && length(newVel[indexP])>0)
		 			r = elast*length(pos - pInter)/(dt*length(newVel[indexP]));
				double3 nInter = approximateNormale_Periodic(pInter,nbFunc,A,k,theta,phi);
				nInter = normalize(nInter);
               			double3 V;
				V.x = newVel[indexP].x - (1+r)*dot(newVel[indexP],nInter)*nInter.x;
				V.y = newVel[indexP].y - (1+r)*dot(newVel[indexP],nInter)*nInter.y;
				V.z = newVel[indexP].z - (1+r)*dot(newVel[indexP],nInter)*nInter.z;
				newVel[indexP] = make_double3(V.x,V.y,V.z);
				newPos[indexP].y = y;
			}
		}
	}
}
/******************************************************************************************/
/******************************************************************************************/
}
/******************************************************************************************/
/******************************************************************************************/
